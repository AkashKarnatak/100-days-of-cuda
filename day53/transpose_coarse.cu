
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hipblas.h>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__device__ __host__ inline size_t cdiv(size_t a, size_t b) {
  return (a + b - 1) / b;
}

struct timer {
  struct timespec start_time, end_time;
};

void start_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->start_time);
}

void stop_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->end_time);
}

double time_diff(struct timer *t) {
  double diff = (t->end_time.tv_sec - t->start_time.tv_sec) +
                (t->end_time.tv_nsec - t->start_time.tv_nsec) / 1000000000.0;
  return diff;
}

struct timer t;

template <const size_t BN, const size_t BM, const size_t CN, const size_t CM>
__global__ void transpose_coarse_kernel(float *in, float *out, size_t N,
                                        size_t M) {
  size_t rowOffset = blockIdx.y * BN;
  size_t colOffset = blockIdx.x * BM;

#pragma unroll 2
  for (size_t innerRow = threadIdx.y; innerRow < BN; innerRow += CN) {
    for (size_t innerCol = threadIdx.x; innerCol < BM; innerCol += CM) {
      out[(colOffset + innerCol) * N + (rowOffset + innerRow)] =
          in[(rowOffset + innerRow) * M + (colOffset + innerCol)];
    }
  }
}

bool allclose(float *A, float *B, size_t N, size_t M) {
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < M; ++j) {
      if (abs(A[i * M + j] - B[i * M + j]) > 1e-4) {
        printf("Mismatch at (%lu, %lu), A = %f and B = %f\n", i, j,
               A[i * M + j], B[i * M + j]);
        return false;
      }
    }
  }
  return true;
}

void print_mat(float *A, size_t N, size_t M) {
  printf("[\n");
  for (size_t row = 0; row < N; ++row) {
    for (size_t col = 0; col < M; ++col) {
      printf("  %f ", A[row * M + col]);
    }
    printf("\n");
  }
  printf("]\n");
}

void transpose_cpu(float *in, float *out, size_t N, size_t M) {
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < M; ++j) {
      out[j * N + i] = in[i * M + j];
    }
  }
}

int main() {
  size_t N, M;
  float *in, *out, *out_base;
  float *in_d, *out_d;
  dim3 numThreads, numBlocks;

  N = M = 16384;

  in = (float *)malloc(N * M * sizeof(float));
  out_base = (float *)malloc(N * M * sizeof(float));
  out = (float *)malloc(N * M * sizeof(float));

  hipMalloc(&in_d, N * M * sizeof(float));
  hipMalloc(&out_d, N * M * sizeof(float));

  for (size_t i = 0; i < N * M; ++i) {
    in[i] = (float)rand() / RAND_MAX;
  }

  transpose_cpu(in, out_base, N, M);

  CUDA_CHECK(
      hipMemcpy(in_d, in, N * M * sizeof(float), hipMemcpyHostToDevice));

  CUDA_CHECK(hipDeviceSynchronize());
  hipMemset(out_d, 0, N * M * sizeof(float));
  start_timer(&t);
  const size_t BN = 16;
  const size_t BM = 16;
  const size_t CN = 8;
  const size_t CM = 16;
  numThreads = dim3(CM, CN);
  numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  transpose_coarse_kernel<BN, BM, CN, CM>
      <<<numBlocks, numThreads>>>(in_d, out_d, N, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  stop_timer(&t);
  CUDA_CHECK(
      hipMemcpy(out, out_d, N * M * sizeof(float), hipMemcpyDeviceToHost));
  printf("Thread coarsened transpose kernel time: %f\n", time_diff(&t));
  printf("Match impl: %s\n\n",
         allclose(out_base, out, N, M) ? "true" : "false");

  hipFree(in_d);
  hipFree(out_d);
  free(in);
  free(out);
  free(out_base);

  return 0;
}
