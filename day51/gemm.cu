
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hipblas.h>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__device__ __host__ inline size_t cdiv(size_t a, size_t b) {
  return (a + b - 1) / b;
}

struct timer {
  struct timespec start_time, end_time;
};

void start_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->start_time);
}

void stop_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->end_time);
}

double time_diff(struct timer *t) {
  double diff = (t->end_time.tv_sec - t->start_time.tv_sec) +
                (t->end_time.tv_nsec - t->start_time.tv_nsec) / 1000000000.0;
  return diff;
}

struct timer t;

template <const size_t BLOCK_SIZE>
__global__ void matmul_naive_kernel(float *A, float *B, float *C, size_t N,
                                    size_t K, size_t M) {
  size_t ty = threadIdx.x / BLOCK_SIZE;
  size_t tx = threadIdx.x % BLOCK_SIZE;
  size_t row = blockIdx.y * BLOCK_SIZE + ty;
  size_t col = blockIdx.x * BLOCK_SIZE + tx;

  if (row >= N && col >= M)
    return;

  float sum = 0.0f;
  for (size_t k = 0; k < K; ++k) {
    sum += A[row * K + k] * B[k * M + col];
  }
  C[row * M + col] = sum;
}

template <const size_t BLOCK_SIZE>
__global__ void matmul_tiled_kernel(float *A, float *B, float *C, size_t N,
                                    size_t K, size_t M) {
  size_t innerRow = threadIdx.x / BLOCK_SIZE;
  size_t innerCol = threadIdx.x % BLOCK_SIZE;
  size_t row = blockIdx.y * BLOCK_SIZE + innerRow;
  size_t col = blockIdx.x * BLOCK_SIZE + innerCol;

  if (row >= N && col >= M)
    return;

  __shared__ float A_s[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B_s[BLOCK_SIZE][BLOCK_SIZE];

  float sum = 0.0f;
  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BLOCK_SIZE) {
    // load data in shared memory
    if (row < N && (tileOffset + innerCol) < M)
      A_s[innerRow][innerCol] = A[row * K + tileOffset + innerCol];
    else
      A_s[innerRow][innerCol] = 0.0f;

    if ((tileOffset + innerRow) < N && col < M)
      B_s[innerRow][innerCol] = B[(tileOffset + innerRow) * M + col];
    else
      B_s[innerRow][innerCol] = 0.0f;

    __syncthreads();

    // compute
    for (size_t k = 0; k < BLOCK_SIZE; ++k) {
      sum += A_s[innerRow][k] * B_s[k][innerCol];
    }

    __syncthreads();
  }

  if (row < N && col < M)
    C[row * M + col] = sum;
}

template <const size_t BN, const size_t BK, const size_t BM, const size_t CN,
          const size_t CM>
__global__ void matmul_tiled_2d_kernel(float *A, float *B, float *C, size_t N,
                                       size_t K, size_t M) {
  const size_t TN = BN / CN;
  const size_t TM = BM / CM;
  const size_t rowAOffset = blockIdx.y * BN;
  const size_t colBOffset = blockIdx.x * BM;
  const size_t rowCOffset = rowAOffset;
  const size_t colCOffset = colBOffset;

  size_t cnt = 0;
  size_t innerColA = threadIdx.x % BK;
  size_t innerRowA = threadIdx.x / BK;
  size_t innerColB = threadIdx.x % BM;
  size_t innerRowB = threadIdx.x / BM;
  size_t innerColC = threadIdx.x % CM;
  size_t innerRowC = threadIdx.x / CM;

  // CN * CM is the number of threads in this block
  size_t strideA = CN * CM / BK;
  size_t strideB = CN * CM / BM;

  __shared__ float A_s[BN][BK];
  __shared__ float B_s[BK][BM];

  float sums[TN * TM] = {0};
  float A_reg[TN], B_reg[TM];

  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BK) {

    // load data in shared memory
    for (size_t innerRowAOffset = 0; innerRowAOffset < BN;
         innerRowAOffset += strideA) {
      if ((rowAOffset + innerRowAOffset + innerRowA) < N &&
          (tileOffset + innerColA) < K)
        A_s[innerRowAOffset + innerRowA][innerColA] =
            A[(rowAOffset + innerRowAOffset + innerRowA) * K + tileOffset +
              innerColA];
      else
        A_s[innerRowAOffset + innerRowA][innerColA] = 0.0f;
    }

    for (size_t innerRowBOffset = 0; innerRowBOffset < BK;
         innerRowBOffset += strideB) {
      if ((tileOffset + innerRowBOffset + innerRowB) < K &&
          (colBOffset + innerColB) < M)
        B_s[innerRowBOffset + innerRowB][innerColB] =
            B[(tileOffset + innerRowBOffset + innerRowB) * M + colBOffset +
              innerColB];
      else
        B_s[innerRowBOffset + innerRowB][innerColB] = 0.0f;
    }

    __syncthreads();

    // compute
    for (size_t k = 0; k < BK; ++k) {
      cnt = 0;

      // load value in registers
      for (size_t i = 0; i < TN; ++i) {
        A_reg[i] = A_s[i * CN + innerRowC][k];
      }
      for (size_t j = 0; j < TM; ++j) {
        B_reg[j] = B_s[k][j * CM + innerColC];
      }

      for (size_t i = 0; i < TN; ++i) {
        for (size_t j = 0; j < TM; ++j) {
          sums[cnt++] += A_reg[i] * B_reg[j];
        }
      }
    }
    __syncthreads();
  }

  cnt = 0;
  for (size_t innerRowCOffset = 0; innerRowCOffset < BN;
       innerRowCOffset += CN) {
    for (size_t innerColCOffset = 0; innerColCOffset < BM;
         innerColCOffset += CM) {
      if ((rowCOffset + innerRowCOffset + innerRowC) < N &&
          (colCOffset + innerColCOffset + innerColC) < M) {
        C[(rowCOffset + innerRowCOffset + innerRowC) * M + colCOffset +
          innerColCOffset + innerColC] = sums[cnt];
      }
      ++cnt;
    }
  }
}

template <const size_t BN, const size_t BK, const size_t BM, const size_t CN,
          const size_t CM>
__global__ void matmul_tiled_vector_kernel(float *A, float *B, float *C,
                                           size_t N, size_t K, size_t M) {
  const size_t TN = BN / CN;
  const size_t TM = BM / CM;
  const size_t rowAOffset = blockIdx.y * BN;
  const size_t colBOffset = blockIdx.x * BM;
  const size_t rowCOffset = rowAOffset;
  const size_t colCOffset = colBOffset;

  size_t cnt = 0;
  size_t innerColA = threadIdx.x % BK;
  size_t innerRowA = threadIdx.x / BK;
  size_t innerColB = threadIdx.x % BM;
  size_t innerRowB = threadIdx.x / BM;
  size_t innerColC = threadIdx.x % CM;
  size_t innerRowC = threadIdx.x / CM;

  // CN * CM is the number of threads in this block
  size_t strideA = CN * CM / BK;
  size_t strideB = CN * CM / BM;

  __shared__ float A_s[BK][BN];
  __shared__ float B_s[BK][BM];

  float sums[TN * TM] = {0};
  float A_reg[TN], B_reg[TM];

  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BK) {

    // load data in shared memory
    for (size_t innerRowAOffset = 0; innerRowAOffset < BN;
         innerRowAOffset += strideA) {
      if ((rowAOffset + innerRowAOffset + innerRowA) < N &&
          (tileOffset + innerColA) < K)
        A_s[innerColA][innerRowAOffset + innerRowA] =
            A[(rowAOffset + innerRowAOffset + innerRowA) * K + tileOffset +
              innerColA];
      else
        A_s[innerColA][innerRowAOffset + innerRowA] = 0.0f;
    }

    for (size_t innerRowBOffset = 0; innerRowBOffset < BK;
         innerRowBOffset += strideB) {
      if ((tileOffset + innerRowBOffset + innerRowB) < K &&
          (colBOffset + innerColB) < M)
        B_s[innerRowBOffset + innerRowB][innerColB] =
            B[(tileOffset + innerRowBOffset + innerRowB) * M + colBOffset +
              innerColB];
      else
        B_s[innerRowBOffset + innerRowB][innerColB] = 0.0f;
    }

    __syncthreads();

    // compute
    for (size_t k = 0; k < BK; ++k) {
      cnt = 0;

      // load value in registers
      for (size_t i = 0; i < TN; ++i) {
        A_reg[i] = A_s[k][i * CN + innerRowC];
      }
      for (size_t j = 0; j < TM; ++j) {
        B_reg[j] = B_s[k][j * CM + innerColC];
      }

      for (size_t i = 0; i < TN; ++i) {
        for (size_t j = 0; j < TM; ++j) {
          sums[cnt++] += A_reg[i] * B_reg[j];
        }
      }
    }
    __syncthreads();
  }

  cnt = 0;
  for (size_t innerRowCOffset = 0; innerRowCOffset < BN;
       innerRowCOffset += CN) {
    for (size_t innerColCOffset = 0; innerColCOffset < BM;
         innerColCOffset += CM) {
      if ((rowCOffset + innerRowCOffset + innerRowC) < N &&
          (colCOffset + innerColCOffset + innerColC) < M) {
        C[(rowCOffset + innerRowCOffset + innerRowC) * M + colCOffset +
          innerColCOffset + innerColC] = sums[cnt];
      }
      ++cnt;
    }
  }
}

bool allclose(float *A, float *B, size_t N, size_t M) {
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < M; ++j) {
      if (abs(A[i * M + j] - B[i * M + j]) > 1e-4) {
        printf("Mismatch at (%lu, %lu), A = %f and B = %f\n", i, j,
               A[i * M + j], B[i * M + j]);
        return false;
      }
    }
  }
  return true;
}

void print_mat(float *A, size_t N, size_t M) {
  printf("[\n");
  for (size_t row = 0; row < N; ++row) {
    for (size_t col = 0; col < M; ++col) {
      printf("  %f ", A[row * M + col]);
    }
    printf("\n");
  }
  printf("]\n");
}

int main() {
  size_t N, K, M;
  float *A, *B, *C_base, *C;
  float *A_d, *B_d, *C_base_d, *C_d;
  dim3 numThreads, numBlocks;

  N = K = M = 4096;

  A = (float *)malloc(N * K * sizeof(float));
  B = (float *)malloc(K * M * sizeof(float));
  C_base = (float *)malloc(N * M * sizeof(float));
  C = (float *)malloc(N * M * sizeof(float));

  hipMalloc(&A_d, N * K * sizeof(float));
  hipMalloc(&B_d, K * M * sizeof(float));
  hipMalloc(&C_base_d, N * M * sizeof(float));
  hipMalloc(&C_d, N * M * sizeof(float));

  for (size_t i = 0; i < N * K; ++i) {
    A[i] = (float)rand() / RAND_MAX;
  }

  for (size_t i = 0; i < K * M; ++i) {
    B[i] = (float)rand() / RAND_MAX;
  }

  CUDA_CHECK(hipMemcpy(A_d, A, N * K * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(B_d, B, K * M * sizeof(float), hipMemcpyHostToDevice));

  CUDA_CHECK(hipDeviceSynchronize());
  const size_t BLOCK_SIZE = 32;
  numThreads = dim3(BLOCK_SIZE * BLOCK_SIZE);
  numBlocks = dim3(cdiv(M, BLOCK_SIZE), cdiv(N, BLOCK_SIZE));
  matmul_naive_kernel<BLOCK_SIZE>
      <<<numBlocks, numThreads>>>(A_d, B_d, C_base_d, N, K, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(C_base, C_base_d, N * M * sizeof(float),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipDeviceSynchronize());
  hipMemset(C_d, 0, N * M * sizeof(float));
  start_timer(&t);
  numThreads = dim3(BLOCK_SIZE * BLOCK_SIZE);
  numBlocks = dim3(cdiv(M, BLOCK_SIZE), cdiv(N, BLOCK_SIZE));
  matmul_naive_kernel<BLOCK_SIZE>
      <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  stop_timer(&t);
  CUDA_CHECK(hipMemcpy(C, C_d, N * M * sizeof(float), hipMemcpyDeviceToHost));
  printf("Naive matmul kernel time: %f\n", time_diff(&t));
  printf("Match impl: %s\n\n", allclose(C_base, C, N, M) ? "true" : "false");

  CUDA_CHECK(hipDeviceSynchronize());
  hipMemset(C_d, 0, N * M * sizeof(float));
  start_timer(&t);
  numThreads = dim3(BLOCK_SIZE * BLOCK_SIZE);
  numBlocks = dim3(cdiv(M, BLOCK_SIZE), cdiv(N, BLOCK_SIZE));
  matmul_tiled_kernel<BLOCK_SIZE>
      <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  stop_timer(&t);
  CUDA_CHECK(hipMemcpy(C, C_d, N * M * sizeof(float), hipMemcpyDeviceToHost));
  printf("Tiled matmul kernel time: %f\n", time_diff(&t));
  printf("Match impl: %s\n\n", allclose(C_base, C, N, M) ? "true" : "false");

  CUDA_CHECK(hipDeviceSynchronize());
  hipMemset(C_d, 0, N * M * sizeof(float));
  const size_t CN = 8;
  const size_t CM = 8;
  const size_t BK = 8;
  const size_t BN = 64;
  const size_t BM = 64;
  assert(BN * BK >= CN * CM &&
         BM * BK >= CN * CM); // number of threads must be less than
  start_timer(&t);
  numThreads = dim3(CN * CM);
  numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  matmul_tiled_2d_kernel<BN, BK, BM, CN, CM>
      <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  stop_timer(&t);
  CUDA_CHECK(hipMemcpy(C, C_d, N * M * sizeof(float), hipMemcpyDeviceToHost));
  printf("Tiled 2d matmul kernel time: %f\n", time_diff(&t));
  printf("Match impl: %s\n\n", allclose(C_base, C, N, M) ? "true" : "false");

  CUDA_CHECK(hipDeviceSynchronize());
  hipMemset(C_d, 0, N * M * sizeof(float));
  assert(BN * BK >= CN * CM &&
         BM * BK >= CN * CM); // number of threads must be less than
  start_timer(&t);
  numThreads = dim3(CN * CM);
  numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  matmul_tiled_vector_kernel<BN, BK, BM, CN, CM>
      <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  stop_timer(&t);
  CUDA_CHECK(hipMemcpy(C, C_d, N * M * sizeof(float), hipMemcpyDeviceToHost));
  printf("Tiled vector matmul kernel time: %f\n", time_diff(&t));
  printf("Match impl: %s\n\n", allclose(C_base, C, N, M) ? "true" : "false");

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_base_d);
  hipFree(C_d);
  free(A);
  free(B);
  free(C_base);
  free(C);

  return 0;
}
