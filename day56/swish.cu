#include <hip/hip_runtime.h>

__device__ __host__ inline size_t cdiv(size_t a, size_t b) {
  return (a + b - 1) / b;
}

__global__ void swish_kernel(const float *__restrict__ in, float *__restrict__ out, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    out[idx] = in[idx] / (1 + expf(-in[idx]));
}

// Note: input, output are all device pointers to float32 arrays
extern "C" void solution(const float* input, float* output, size_t n, size_t m) {
    size_t N = m, M = n;
    dim3 numThreads(256);
    dim3 numBlocks(cdiv(N * M, numThreads.x));
    swish_kernel<<<numBlocks, numThreads>>>(input, output, N * M);
}
