
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define BLOCK_DIM 1024

inline size_t cdiv(size_t a, size_t b) { return (a + b - 1) / b; }

struct timer {
  struct timespec start_time, end_time;
};

void start_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->start_time);
}

void stop_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->end_time);
}

double time_diff(struct timer *t) {
  double diff = (t->end_time.tv_sec - t->start_time.tv_sec) +
                (t->end_time.tv_nsec - t->start_time.tv_nsec) / 1000000000.0;
  return diff;
}

struct timer t;

__global__ void brent_kung_inclusive_kernel(float *in, float *out,
                                            float *partial_sums, size_t N) {
  size_t offset = blockIdx.x * blockDim.x * 2;
  size_t tid = threadIdx.x;

  __shared__ float sram[BLOCK_DIM * 2];

  if (offset + tid < N)
    sram[tid] = in[offset + tid];
  else
    sram[tid] = 0.0f;
  if (offset + BLOCK_DIM + tid < N)
    sram[BLOCK_DIM + tid] = in[offset + BLOCK_DIM + tid];
  else
    sram[tid + BLOCK_DIM] = 0.0f;

  __syncthreads();

  // reduction step
  for (size_t s = 1; s <= BLOCK_DIM; s *= 2) {
    size_t j = (tid + 1) * 2 * s - 1;
    if (j < 2 * BLOCK_DIM)
      sram[j] += sram[j - s];
    __syncthreads();
  }

  // post reduction step
  for (size_t s = BLOCK_DIM / 2; s > 0; s /= 2) {
    size_t j = (tid + 1) * 2 * s - 1;
    if (j + s < 2 * BLOCK_DIM)
      sram[j + s] += sram[j];
    __syncthreads();
  }

  if (offset + tid < N)
    out[offset + tid] = sram[tid];
  if (offset + BLOCK_DIM + tid < N)
    out[offset + BLOCK_DIM + tid] = sram[BLOCK_DIM + tid];

  if (tid == BLOCK_DIM - 1) {
    partial_sums[blockIdx.x] = sram[2 * BLOCK_DIM - 1];
  }
}

__global__ void brent_kung_exclusive_kernel(float *in, float *out,
                                            float *partial_sums, size_t N) {
  size_t offset = blockIdx.x * blockDim.x * 2;
  size_t tid = threadIdx.x;

  __shared__ float sram[BLOCK_DIM * 2];

  if (tid > 0 && offset + tid < N)
    sram[tid] = in[offset + tid - 1];
  else
    sram[tid] = 0.0f;
  if (offset + BLOCK_DIM + tid < N)
    sram[BLOCK_DIM + tid] = in[offset + BLOCK_DIM + tid - 1];
  else
    sram[tid + BLOCK_DIM] = 0.0f;

  __syncthreads();

  // reduction step
  for (size_t s = 1; s <= BLOCK_DIM; s *= 2) {
    size_t j = (tid + 1) * 2 * s - 1;
    if (j < 2 * BLOCK_DIM)
      sram[j] += sram[j - s];
    __syncthreads();
  }

  // post reduction step
  for (size_t s = BLOCK_DIM / 2; s > 0; s /= 2) {
    size_t j = (tid + 1) * 2 * s - 1;
    if (j + s < 2 * BLOCK_DIM)
      sram[j + s] += sram[j];
    __syncthreads();
  }

  if (offset + tid < N)
    out[offset + tid] = sram[tid];
  if (offset + BLOCK_DIM + tid < N)
    out[offset + BLOCK_DIM + tid] = sram[BLOCK_DIM + tid];

  if (tid == BLOCK_DIM - 1) {
    float last = N - 1 <= offset + BLOCK_DIM + tid
                     ? in[N - 1]
                     : in[offset + BLOCK_DIM + tid];
    partial_sums[blockIdx.x] = sram[2 * BLOCK_DIM - 1] + last;
  }
}

__global__ void add_inclusive_kernel(float *out, float *partial_sums,
                                     size_t N) {
  size_t offset = blockIdx.x * blockDim.x * 2;
  size_t tid = threadIdx.x;

  if (blockIdx.x == 0)
    return;

  if (offset + tid < N)
    out[offset + tid] += partial_sums[blockIdx.x - 1];
  if (offset + BLOCK_DIM + tid < N)
    out[offset + BLOCK_DIM + tid] += partial_sums[blockIdx.x - 1];
}

__global__ void add_exclusive_kernel(float *out, float *partial_sums,
                                     size_t N) {
  size_t offset = blockIdx.x * blockDim.x * 2;
  size_t tid = threadIdx.x;

  if (blockIdx.x == 0)
    return;

  if (offset + tid < N)
    out[offset + tid] += partial_sums[blockIdx.x];
  if (offset + BLOCK_DIM + tid < N)
    out[offset + BLOCK_DIM + tid] += partial_sums[blockIdx.x];
}

void brent_kung_inclusive_gpu_d(float *in_d, float *out_d, size_t N) {
  float *partial_sums_d;

  size_t numThreads = BLOCK_DIM;
  size_t numBlocks = cdiv(N, numThreads * 2);

  hipMalloc(&partial_sums_d, numBlocks * sizeof(float));

  brent_kung_inclusive_kernel<<<numBlocks, numThreads>>>(in_d, out_d,
                                                         partial_sums_d, N);
  hipDeviceSynchronize();

  if (numBlocks > 1)
    brent_kung_inclusive_gpu_d(partial_sums_d, partial_sums_d, numBlocks);

  add_inclusive_kernel<<<numBlocks, numThreads>>>(out_d, partial_sums_d, N);
  hipDeviceSynchronize();

  hipFree(partial_sums_d);
}

void brent_kung_exclusive_gpu_d(float *in_d, float *out_d, size_t N) {
  float *partial_sums_d;

  size_t numThreads = BLOCK_DIM;
  size_t numBlocks = cdiv(N, numThreads * 2);

  hipMalloc(&partial_sums_d, numBlocks * sizeof(float));

  brent_kung_exclusive_kernel<<<numBlocks, numThreads>>>(in_d, out_d,
                                                         partial_sums_d, N);
  hipDeviceSynchronize();

  if (numBlocks > 1)
    brent_kung_exclusive_gpu_d(partial_sums_d, partial_sums_d, numBlocks);

  add_exclusive_kernel<<<numBlocks, numThreads>>>(out_d, partial_sums_d, N);
  hipDeviceSynchronize();

  hipFree(partial_sums_d);
}

void brent_kung_inclusive_gpu(float *in, float *out, size_t N) {
  float *in_d, *out_d;

  hipMalloc(&in_d, N * sizeof(float));
  hipMalloc(&out_d, N * sizeof(float));

  hipMemcpy(in_d, in, N * sizeof(float), hipMemcpyHostToDevice);

  start_timer(&t);
  brent_kung_inclusive_gpu_d(in_d, out_d, N);
  stop_timer(&t);
  printf("GPU time: %f\n", time_diff(&t));

  hipMemcpy(out, out_d, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(in_d);
  hipFree(out_d);
}

void brent_kung_exclusive_gpu(float *in, float *out, size_t N) {
  float *in_d, *out_d;

  hipMalloc(&in_d, N * sizeof(float));
  hipMalloc(&out_d, N * sizeof(float));

  hipMemcpy(in_d, in, N * sizeof(float), hipMemcpyHostToDevice);

  start_timer(&t);
  brent_kung_exclusive_gpu_d(in_d, out_d, N);
  stop_timer(&t);
  printf("GPU time: %f\n", time_diff(&t));

  hipMemcpy(out, out_d, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(in_d);
  hipFree(out_d);
}

void brent_kung_inclusive_cpu(float *in, float *out, size_t N) {
  start_timer(&t);
  out[0] = in[0];
  for (size_t i = 1; i < N; ++i) {
    out[i] = out[i - 1] + in[i];
  }
  stop_timer(&t);
  printf("CPU time: %f\n", time_diff(&t));
}

void brent_kung_exclusive_cpu(float *in, float *out, size_t N) {
  start_timer(&t);
  out[0] = 0;
  for (size_t i = 1; i < N; ++i) {
    out[i] = out[i - 1] + in[i - 1];
  }
  stop_timer(&t);
  printf("CPU time: %f\n", time_diff(&t));
}

bool allclose(float *A, float *B, size_t N) {
  for (size_t i = 0; i < N; ++i) {
    if (abs(A[i] - B[i]) > 6) {
      printf("Mismatch at (%ld,): A = %f and B = %f\n", i, A[i], B[i]);
      return false;
    }
  }
  return true;
}

void print(float *A, size_t N) {
  printf("[\n");
  for (size_t i = 0; i < N; ++i) {
    printf(" %f", A[i]);
  }
  printf("\n]\n");
}

int main() {
  size_t N;
  float *in, *out_cpu, *out_gpu;

  N = 1'000'000;
  in = (float *)malloc(N * sizeof(float));
  out_cpu = (float *)malloc(N * sizeof(float));
  out_gpu = (float *)malloc(N * sizeof(float));

  for (size_t i = 0; i < N; ++i) {
    in[i] = (float)rand() / RAND_MAX;
    // in[i] = 0.1;
  }

  brent_kung_exclusive_cpu(in, out_cpu, N);

  brent_kung_exclusive_gpu(in, out_gpu, N);

  // print(out_cpu, N);
  // print(out_gpu, N);

  printf("Match impl: %s\n", allclose(out_cpu, out_gpu, N) ? "true" : "false");

  return 0;
}
