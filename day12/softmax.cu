
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define TILE_SIZE 1024

struct timer {
  struct timespec start_time, end_time;
};

void start_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->start_time);
}

void stop_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->end_time);
}

double time_diff(struct timer *t) {
  double diff = (t->end_time.tv_sec - t->start_time.tv_sec) +
                (t->end_time.tv_nsec - t->start_time.tv_nsec) / 1000000000.0;
  return diff;
}

struct timer t;

__host__ __device__ size_t cdiv(size_t a, size_t b) { return (a + b - 1) / b; }

__global__ void naive_softmax_kernel(float *in_data, float *out_data, size_t N,
                                     size_t M) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= N)
    return;

  float sum, maximum;

  maximum = -INFINITY;
  for (size_t k = 0; k < M; ++k) {
    if (in_data[i * M + k] > maximum)
      maximum = in_data[i * M + k];
  }

  sum = 0.0f;
  for (size_t k = 0; k < M; ++k) {
    sum += expf(in_data[i * M + k] - maximum);
  }

  for (size_t k = 0; k < M; ++k) {
    out_data[i * M + k] = expf(in_data[i * M + k] - maximum) / sum;
  }
}

__global__ void online_softmax_kernel(float *in_data, float *out_data, size_t N,
                                      size_t M) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= N)
    return;

  float sum, maximum;

  maximum = -INFINITY;
  for (size_t k = 0; k < M; ++k) {
    float curr = in_data[i * M + k];
    if (curr > maximum) {
      sum *= expf(maximum - curr);
      maximum = curr;
    }
    sum += expf(curr - maximum);
  }

  for (size_t k = 0; k < M; ++k) {
    out_data[i * M + k] = expf(in_data[i * M + k] - maximum) / sum;
  }
}

// // I came up with this weird algo (performs worse than naive softmax)
// __global__ void tiled_softmax_kernel(float *in_data, float *out_data, size_t
// N,
//                                      size_t M) {
//   size_t row = blockIdx.x;
//
//   __shared__ float in_data_s[TILE_SIZE];
//   __shared__ float sum_s, maximum_s;
//
//   sum_s = 0.0f, maximum_s = -INFINITY;
//   for (size_t tileIdx = 0; tileIdx < cdiv(M, TILE_SIZE); ++tileIdx) {
//     if (tileIdx * TILE_SIZE + threadIdx.x < M)
//       in_data_s[threadIdx.x] =
//           in_data[row * M + tileIdx * TILE_SIZE + threadIdx.x];
//     else
//       in_data_s[threadIdx.x] = -INFINITY;
//
//     __syncthreads();
//
//     if (threadIdx.x == 0) {
//       float sum = sum_s, maximum = maximum_s;
//       for (size_t i = 0; i < TILE_SIZE; ++i) {
//         if (tileIdx * TILE_SIZE + i >= M)
//           continue;
//         float curr = in_data_s[i];
//         if (curr > maximum) {
//           sum *= expf(maximum - curr);
//           maximum = curr;
//         }
//         sum += expf(curr - maximum);
//       }
//       sum_s = sum + sum_s * expf(maximum_s - maximum),
//       maximum_s = max(maximum_s, maximum);
//     }
//
//     __syncthreads();
//   }
//
//   for (size_t tileIdx = 0; tileIdx < cdiv(M, TILE_SIZE); ++tileIdx) {
//     if (tileIdx * TILE_SIZE + threadIdx.x < M)
//       out_data[row * M + tileIdx * TILE_SIZE + threadIdx.x] =
//           expf(in_data[row * M + tileIdx * TILE_SIZE + threadIdx.x] -
//                maximum_s) /
//           sum_s;
//   }
// }

__global__ void block_reduction_softmax_kernel(float *in_data, float *out_data, size_t N,
                                     size_t M) {
  size_t row = blockIdx.x;

  float local_sum, local_max, global_sum, global_max;
  __shared__ float mem_s[TILE_SIZE];

  local_sum = 0.0f, local_max = -INFINITY;
  for (size_t tileIdx = 0; tileIdx < cdiv(M, TILE_SIZE); ++tileIdx) {
    if (tileIdx * TILE_SIZE + threadIdx.x >= M)
      continue;
    float curr = in_data[row * M + tileIdx * TILE_SIZE + threadIdx.x];
    if (curr > local_max) {
      local_sum *= expf(local_max - curr);
      local_max = curr;
    }
    local_sum += expf(curr - local_max);
  }

  mem_s[threadIdx.x] = local_max;

  __syncthreads();

  for (size_t numThreads = TILE_SIZE / 2; numThreads > 0; numThreads /= 2) {
    if (threadIdx.x < numThreads) {
      mem_s[threadIdx.x] =
          max(mem_s[threadIdx.x], mem_s[threadIdx.x + numThreads]);
    }
    __syncthreads();
  }

  global_max = mem_s[0];

  mem_s[threadIdx.x] = local_sum * expf(local_max - global_max);

  __syncthreads();

  for (size_t numThreads = TILE_SIZE / 2; numThreads > 0; numThreads /= 2) {
    if (threadIdx.x < numThreads) {
      mem_s[threadIdx.x] += mem_s[threadIdx.x + numThreads];
    }
    __syncthreads();
  }

  global_sum = mem_s[0];

  __syncthreads();

  for (size_t tileIdx = 0; tileIdx < cdiv(M, TILE_SIZE); ++tileIdx) {
    if (tileIdx * TILE_SIZE + threadIdx.x < M) {
      out_data[row * M + tileIdx * TILE_SIZE + threadIdx.x] =
          expf(in_data[row * M + tileIdx * TILE_SIZE + threadIdx.x] -
               global_max) /
          global_sum;
    }
  }
}

void naive_softmax_gpu(float *in_data, float *out_data, size_t N, size_t M) {
  float *in_data_d, *out_data_d;

  // allocate memory on GPU
  hipMalloc(&in_data_d, N * M * sizeof(float));
  hipMalloc(&out_data_d, N * M * sizeof(float));

  // copy data from host to device
  hipMemcpy(in_data_d, in_data, N * M * sizeof(float), hipMemcpyHostToDevice);

  // perform computation
  size_t numThreads = 1024;
  size_t numBlocks = cdiv(N, numThreads);
  start_timer(&t);
  hipDeviceSynchronize();
  naive_softmax_kernel<<<numBlocks, numThreads>>>(in_data_d, out_data_d, N, M);
  hipDeviceSynchronize();
  stop_timer(&t);
  printf("GPU time (Row wise): %f\n", time_diff(&t));

  // copy data from device to host
  hipMemcpy(out_data, out_data_d, N * M * sizeof(float),
             hipMemcpyDeviceToHost);

  // free memory
  hipFree(in_data_d);
  hipFree(out_data_d);
}

void online_softmax_gpu(float *in_data, float *out_data, size_t N, size_t M) {
  float *in_data_d, *out_data_d;

  // allocate memory on GPU
  hipMalloc(&in_data_d, N * M * sizeof(float));
  hipMalloc(&out_data_d, N * M * sizeof(float));

  // copy data from host to device
  hipMemcpy(in_data_d, in_data, N * M * sizeof(float), hipMemcpyHostToDevice);

  // perform computation
  size_t numThreads = 1024;
  size_t numBlocks = cdiv(N, numThreads);
  start_timer(&t);
  hipDeviceSynchronize();
  online_softmax_kernel<<<numBlocks, numThreads>>>(in_data_d, out_data_d, N, M);
  hipDeviceSynchronize();
  stop_timer(&t);
  printf("GPU time (2D): %f\n", time_diff(&t));

  // copy data from device to host
  hipMemcpy(out_data, out_data_d, N * M * sizeof(float),
             hipMemcpyDeviceToHost);

  // free memory
  hipFree(in_data_d);
  hipFree(out_data_d);
}

void block_reduction_softmax_gpu(float *in_data, float *out_data, size_t N, size_t M) {
  float *in_data_d, *out_data_d;

  // allocate memory on GPU
  hipMalloc(&in_data_d, N * M * sizeof(float));
  hipMalloc(&out_data_d, N * M * sizeof(float));

  // copy data from host to device
  hipMemcpy(in_data_d, in_data, N * M * sizeof(float), hipMemcpyHostToDevice);

  // perform computation
  size_t numThreads = 1024;
  size_t numBlocks = N;
  start_timer(&t);
  hipDeviceSynchronize();
  block_reduction_softmax_kernel<<<numBlocks, numThreads>>>(in_data_d, out_data_d, N, M);
  hipDeviceSynchronize();
  stop_timer(&t);
  printf("GPU time (Block reduction): %f\n", time_diff(&t));

  // copy data from device to host
  hipMemcpy(out_data, out_data_d, N * M * sizeof(float),
             hipMemcpyDeviceToHost);

  // free memory
  hipFree(in_data_d);
  hipFree(out_data_d);
}

void softmax_cpu(float *pred_data, float *out_data, size_t N, size_t M) {
  start_timer(&t);
  for (size_t row = 0; row < N; ++row) {
    float sum, maximum;

    maximum = -INFINITY;
    for (size_t col = 0; col < M; ++col) {
      if (pred_data[row * M + col] > maximum) {
        maximum = pred_data[row * M + col];
      }
    }

    sum = 0.0f;
    for (size_t col = 0; col < M; ++col) {
      sum += expf(pred_data[row * M + col] - maximum);
    }

    for (size_t col = 0; col < M; ++col) {
      out_data[row * M + col] = expf(pred_data[row * M + col] - maximum) / sum;
    }
  }
  stop_timer(&t);
  printf("CPU time: %f\n", time_diff(&t));
}

bool allclose(float *a, float *b, size_t N) {
  for (size_t i = 0; i < N; ++i) {
    if (abs(a[i] - b[i]) > 1e-4)
      return false;
  }
  return true;
}

int32_t main() {
  hipDeviceSynchronize();

  float *in_data, *out_data_cpu, *out_data_gpu;
  size_t N, M;

  N = 32786, M = 1024;

  in_data = (float *)malloc(N * M * sizeof(float));
  out_data_cpu = (float *)malloc(N * M * sizeof(float));
  out_data_gpu = (float *)malloc(N * M * sizeof(float));

  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < M; ++j) {
      in_data[i * M + j] = (float)rand() / RAND_MAX;
    }
  }

  softmax_cpu(in_data, out_data_cpu, N, M);

  naive_softmax_gpu(in_data, out_data_gpu, N, M);

  printf("CPU and GPU match(Naive): %s\n",
         allclose(out_data_cpu, out_data_gpu, N * M) ? "true" : "false");

  online_softmax_gpu(in_data, out_data_gpu, N, M);

  printf("CPU and GPU match (Online): %s\n",
         allclose(out_data_cpu, out_data_gpu, N * M) ? "true" : "false");

  block_reduction_softmax_gpu(in_data, out_data_gpu, N, M);

  printf("CPU and GPU match (Block Reduction): %s\n",
         allclose(out_data_cpu, out_data_gpu, N * M) ? "true" : "false");

  return 0;
}
