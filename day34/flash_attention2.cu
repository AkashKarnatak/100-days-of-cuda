
#include <hip/hip_runtime.h>
#include <stdio.h>

size_t cdiv(size_t a, size_t b) { return (a + b - 1) / b; }

__global__ void flash_attention_kernel(float *query, float *key, float *value,
                                       float *output, float *sums, float *maxes,
                                       size_t N, size_t d, size_t Br, size_t Bc,
                                       size_t Tr, size_t Tc) {
  size_t j = blockIdx.x;
  size_t tid = threadIdx.x;
  float *Q, *K, *V, *S;

  extern __shared__ float sram[];
  Q = &sram[0];                   // Br x d
  K = &sram[Br * d];              // Bc x d
  V = &sram[Br * d + Bc * d];     // Bc x d
  S = &sram[Br * d + 2 * Bc * d]; // Br x Bc

  __syncthreads();

  // load Q
  for (size_t k = 0; k < d; ++k) {
    if (j * Br + tid < N) {
      Q[tid * d + k] = query[(j * Br + tid) * d + k];
    } else {
      Q[tid * d + k] = 0;
    }
  }

  __syncthreads();

  float m_prev, l_prev;

  if (j * Br + tid < N)
    m_prev = maxes[j * Br + tid], l_prev = sums[j * Br + tid];
  else
    m_prev = -INFINITY, l_prev = 0.0;

  for (size_t i = 0; i < Tc; ++i) {
    __syncthreads();

    // load Kj, Vj
    for (size_t k = 0; k < d; ++k) {
      if (i * Bc + tid < N) {
        K[tid * d + k] = key[(i * Bc + tid) * d + k];
        V[tid * d + k] = value[(i * Bc + tid) * d + k];
      } else {
        K[tid * d + k] = 0;
        V[tid * d + k] = 0;
      }
    }

    __syncthreads();

    if (j * Br + tid >= N)
      continue;

    float m_curr, l_curr, m_new, l_new;

    m_curr = -INFINITY, l_curr = 0;

    for (size_t col = 0; col < Bc; ++col) {
      float sum = 0.0f;
      for (size_t k = 0; k < d; ++k) {
        sum += Q[tid * d + k] * K[col * d + k];
      }
      if (i * Bc + col < N)
        S[tid * Bc + col] = sum / sqrtf(d);
      else
        S[tid * Bc + col] = -INFINITY;
      m_curr = fmaxf(m_curr, sum / sqrtf(d));
    }

    m_new = max(m_prev, m_curr);

    for (size_t col = 0; col < Bc; ++col) {
      S[tid * Bc + col] = expf(S[tid * Bc + col] - m_new);
      l_curr += S[tid * Bc + col];
    }

    l_new = l_prev * expf(m_prev - m_new) + l_curr;

    for (size_t col = 0; col < d; ++col) {
      float sum = 0.0f;
      for (size_t k = 0; k < Bc; ++k) {
        sum += S[tid * Bc + k] * V[k * d + col];
      }
      output[(j * Br + tid) * d + col] =
          output[(j * Br + tid) * d + col] * expf(m_prev - m_new) + sum;
    }
    m_prev = m_new;
    l_prev = l_new;
  }

  if (j * Br + tid < N) {
    for (size_t col = 0; col < d; ++col)
      output[(j * Br + tid) * d + col] /= l_prev;
    sums[j * Br + tid] = l_prev;
    maxes[j * Br + tid] = m_prev;
  }
}

extern "C" {
void flash_attention_gpu(float *query, float *key, float *value, float *output,
                         float *sums, float *maxes, size_t N, size_t d,
                         size_t Br, size_t Bc) {
  size_t Tr, Tc;
  float *query_d, *key_d, *value_d, *output_d, *sums_d, *maxes_d;

  Tr = cdiv(N, Br), Tc = cdiv(N, Bc);

  hipMalloc(&query_d, (N * d) * sizeof(float));
  hipMalloc(&key_d, (N * d) * sizeof(float));
  hipMalloc(&value_d, (N * d) * sizeof(float));
  hipMalloc(&output_d, (N * d) * sizeof(float));
  hipMalloc(&sums_d, N * sizeof(float));
  hipMalloc(&maxes_d, N * sizeof(float));

  hipMemcpy(query_d, query, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(key_d, key, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(value_d, value, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(output_d, output, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(sums_d, sums, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(maxes_d, maxes, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 numThreads(Br);
  dim3 numBlocks(Tr);
  size_t sramMem = (Br * d    // Q
                    + Bc * d  // K
                    + Bc * d  // V
                    + Br * Bc // S
                    ) *
                   sizeof(float);
  flash_attention_kernel<<<numBlocks, numThreads, sramMem>>>(
      query_d, key_d, value_d, output_d, sums_d, maxes_d, N, d, Br, Bc, Tr, Tc);
  hipError_t err = hipGetLastError(); // Check for launch errors
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, (N * d) * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(sums, sums_d, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(maxes, maxes_d, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(query_d);
  hipFree(key_d);
  hipFree(value_d);
  hipFree(output_d);
  hipFree(sums_d);
  hipFree(maxes_d);
}
}

int main() {
  hipDeviceSynchronize();

  size_t N, d, Br, Bc;
  float *query, *key, *value, *output, *sums, *maxes;

  N = 3276, d = 128;
  Br = 24, Bc = 24;

  query = (float *)malloc((N * d) * sizeof(float));
  key = (float *)malloc((N * d) * sizeof(float));
  value = (float *)malloc((N * d) * sizeof(float));
  output = (float *)malloc((N * d) * sizeof(float));
  sums = (float *)malloc(N * sizeof(float));
  maxes = (float *)malloc(N * sizeof(float));

  for (size_t i = 0; i < N * d; ++i) {
    query[i] = (float)rand() / RAND_MAX;
    key[i] = (float)rand() / RAND_MAX;
    value[i] = (float)rand() / RAND_MAX;
    output[i] = 0;
  }

  for (size_t i = 0; i < N; ++i) {
    sums[i] = 0;
    maxes[i] = -INFINITY;
  }

  flash_attention_gpu(query, key, value, output, sums, maxes, N, d, Br, Bc);

  // for (size_t i = 0; i < N; ++i) {
  //   for (size_t j = 0; j < d; ++j) {
  //     printf(" %.4f", output[i * d + j]);
  //   }
  //   printf("\n");
  // }

  free(query);
  free(key);
  free(value);
  free(output);
  free(sums);
  free(maxes);
}
