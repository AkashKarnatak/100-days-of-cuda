
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hipblas.h>
#include <stdio.h>
#include <time.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__device__ __host__ inline size_t cdiv(size_t a, size_t b) {
  return (a + b - 1) / b;
}

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

#define BENCHMARK_CUDA_KERNEL(name, warmup_iters, bench_iters, kernel_call)    \
  do {                                                                         \
    printf("Benchmarking %s...\n", name);                                      \
    /* warmup */                                                               \
    for (int _i = 0; _i < warmup_iters; ++_i) {                                \
      kernel_call;                                                             \
    }                                                                          \
    CUDA_CHECK(hipDeviceSynchronize());                                       \
                                                                               \
    hipEvent_t _start, _stop;                                                 \
    CUDA_CHECK(hipEventCreate(&_start));                                      \
    CUDA_CHECK(hipEventCreate(&_stop));                                       \
                                                                               \
    /* benchmark */                                                            \
    CUDA_CHECK(hipEventRecord(_start));                                       \
    for (int _i = 0; _i < bench_iters; ++_i) {                                 \
      kernel_call;                                                             \
    }                                                                          \
    CUDA_CHECK(hipEventRecord(_stop));                                        \
    CUDA_CHECK(hipEventSynchronize(_stop));                                   \
                                                                               \
    float _elapsed_ms = 0.0f;                                                  \
    CUDA_CHECK(hipEventElapsedTime(&_elapsed_ms, _start, _stop));             \
                                                                               \
    printf("Average runtime for %s: %.4f ms\n\n", name,                        \
           _elapsed_ms / bench_iters);                                         \
                                                                               \
    CUDA_CHECK(hipEventDestroy(_start));                                      \
    CUDA_CHECK(hipEventDestroy(_stop));                                       \
  } while (0)

template <const size_t BLOCK_SIZE>
__global__ void matmul_naive_kernel(float *A, float *B, float *C, size_t N,
                                    size_t K, size_t M) {
  size_t ty = threadIdx.x / BLOCK_SIZE;
  size_t tx = threadIdx.x % BLOCK_SIZE;
  size_t row = blockIdx.y * BLOCK_SIZE + ty;
  size_t col = blockIdx.x * BLOCK_SIZE + tx;

  if (row >= N && col >= M)
    return;

  float sum = 0.0f;
  for (size_t k = 0; k < K; ++k) {
    sum += A[row * K + k] * B[k * M + col];
  }
  C[row * M + col] = sum;
}

template <const size_t BLOCK_SIZE>
__global__ void matmul_tiled_kernel(float *A, float *B, float *C, size_t N,
                                    size_t K, size_t M) {
  size_t innerRow = threadIdx.x / BLOCK_SIZE;
  size_t innerCol = threadIdx.x % BLOCK_SIZE;
  size_t row = blockIdx.y * BLOCK_SIZE + innerRow;
  size_t col = blockIdx.x * BLOCK_SIZE + innerCol;

  if (row >= N && col >= M)
    return;

  __shared__ float A_s[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B_s[BLOCK_SIZE][BLOCK_SIZE];

  float sum = 0.0f;
  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BLOCK_SIZE) {
    // load data in shared memory
    if (row < N && (tileOffset + innerCol) < M)
      A_s[innerRow][innerCol] = A[row * K + tileOffset + innerCol];
    else
      A_s[innerRow][innerCol] = 0.0f;

    if ((tileOffset + innerRow) < N && col < M)
      B_s[innerRow][innerCol] = B[(tileOffset + innerRow) * M + col];
    else
      B_s[innerRow][innerCol] = 0.0f;

    __syncthreads();

    // compute
    for (size_t k = 0; k < BLOCK_SIZE; ++k) {
      sum += A_s[innerRow][k] * B_s[k][innerCol];
    }

    __syncthreads();
  }

  if (row < N && col < M)
    C[row * M + col] = sum;
}

template <const size_t BN, const size_t BK, const size_t BM, const size_t CN,
          const size_t CM>
__global__ void matmul_tiled_2d_kernel(float *A, float *B, float *C, size_t N,
                                       size_t K, size_t M) {
  const size_t TN = BN / CN;
  const size_t TM = BM / CM;
  const size_t rowAOffset = blockIdx.y * BN;
  const size_t colBOffset = blockIdx.x * BM;
  const size_t rowCOffset = rowAOffset;
  const size_t colCOffset = colBOffset;

  size_t cnt = 0;
  size_t innerColA = threadIdx.x % BK;
  size_t innerRowA = threadIdx.x / BK;
  size_t innerColB = threadIdx.x % BM;
  size_t innerRowB = threadIdx.x / BM;
  size_t innerColC = threadIdx.x % CM;
  size_t innerRowC = threadIdx.x / CM;

  // CN * CM is the number of threads in this block
  size_t strideA = CN * CM / BK;
  size_t strideB = CN * CM / BM;

  __shared__ float A_s[BN][BK];
  __shared__ float B_s[BK][BM];

  float sums[TN * TM] = {0};
  float A_reg[TN], B_reg[TM];

  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BK) {

    // load data in shared memory
    for (size_t innerRowAOffset = 0; innerRowAOffset < BN;
         innerRowAOffset += strideA) {
      if ((rowAOffset + innerRowAOffset + innerRowA) < N &&
          (tileOffset + innerColA) < K)
        A_s[innerRowAOffset + innerRowA][innerColA] =
            A[(rowAOffset + innerRowAOffset + innerRowA) * K + tileOffset +
              innerColA];
      else
        A_s[innerRowAOffset + innerRowA][innerColA] = 0.0f;
    }

    for (size_t innerRowBOffset = 0; innerRowBOffset < BK;
         innerRowBOffset += strideB) {
      if ((tileOffset + innerRowBOffset + innerRowB) < K &&
          (colBOffset + innerColB) < M)
        B_s[innerRowBOffset + innerRowB][innerColB] =
            B[(tileOffset + innerRowBOffset + innerRowB) * M + colBOffset +
              innerColB];
      else
        B_s[innerRowBOffset + innerRowB][innerColB] = 0.0f;
    }

    __syncthreads();

    // compute
    for (size_t k = 0; k < BK; ++k) {
      cnt = 0;

      // load value in registers
      for (size_t i = 0; i < TN; ++i) {
        A_reg[i] = A_s[i * CN + innerRowC][k];
      }
      for (size_t j = 0; j < TM; ++j) {
        B_reg[j] = B_s[k][j * CM + innerColC];
      }

      for (size_t i = 0; i < TN; ++i) {
        for (size_t j = 0; j < TM; ++j) {
          sums[cnt++] += A_reg[i] * B_reg[j];
        }
      }
    }
    __syncthreads();
  }

  cnt = 0;
  for (size_t innerRowCOffset = 0; innerRowCOffset < BN;
       innerRowCOffset += CN) {
    for (size_t innerColCOffset = 0; innerColCOffset < BM;
         innerColCOffset += CM) {
      if ((rowCOffset + innerRowCOffset + innerRowC) < N &&
          (colCOffset + innerColCOffset + innerColC) < M) {
        C[(rowCOffset + innerRowCOffset + innerRowC) * M + colCOffset +
          innerColCOffset + innerColC] = sums[cnt];
      }
      ++cnt;
    }
  }
}

template <const size_t BN, const size_t BK, const size_t BM, const size_t CN,
          const size_t CM>
__global__ void matmul_tiled_vector_kernel(float *A, float *B, float *C,
                                           size_t N, size_t K, size_t M) {
  const size_t TN = BN / CN;
  const size_t TM = BM / CM;
  const size_t rowAOffset = blockIdx.y * BN;
  const size_t colBOffset = blockIdx.x * BM;
  const size_t rowCOffset = rowAOffset;
  const size_t colCOffset = colBOffset;

  size_t cnt = 0;
  size_t innerColA = threadIdx.x % (BK / 4);
  size_t innerRowA = threadIdx.x / (BK / 4);
  size_t innerColB = threadIdx.x % (BM / 4);
  size_t innerRowB = threadIdx.x / (BM / 4);
  size_t innerColC = threadIdx.x % CM;
  size_t innerRowC = threadIdx.x / CM;

  // CN * CM is the number of threads in this block
  size_t strideA = CN * CM / (BK / 4);
  size_t strideB = CN * CM / (BM / 4);

  __shared__ float A_s[BK][BN];
  __shared__ float B_s[BK][BM + 5];

  float sums[TN * TM] = {0};
  float A_reg[TN], B_reg[TM];

  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BK) {

    // load data in shared memory
    for (size_t innerRowAOffset = 0; innerRowAOffset < BN;
         innerRowAOffset += strideA) {
      if ((rowAOffset + innerRowAOffset + innerRowA) < N &&
          (tileOffset + innerColA * 4) < K) {
        float4 tmp =
            *((float4 *)&A[(rowAOffset + innerRowAOffset + innerRowA) * K +
                           tileOffset + innerColA * 4]);
        // TODO fix bank conflicts
        A_s[innerColA * 4 + 0][innerRowAOffset + innerRowA] = tmp.x;
        A_s[innerColA * 4 + 1][innerRowAOffset + innerRowA] = tmp.y;
        A_s[innerColA * 4 + 2][innerRowAOffset + innerRowA] = tmp.z;
        A_s[innerColA * 4 + 3][innerRowAOffset + innerRowA] = tmp.w;
      } else {
        A_s[innerColA * 4 + 0][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 1][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 2][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 3][innerRowAOffset + innerRowA] = 0.0f;
      }
    }

    for (size_t innerRowBOffset = 0; innerRowBOffset < BK;
         innerRowBOffset += strideB) {
      if ((tileOffset + innerRowBOffset + innerRowB) < K &&
          (colBOffset + innerColB * 4) < M) {
        float4 tmp =
            *(float4 *)&B[(tileOffset + innerRowBOffset + innerRowB) * M +
                          colBOffset + innerColB * 4];
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 0] = tmp.x;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 1] = tmp.y;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 2] = tmp.z;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 3] = tmp.w;
      } else {
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 0] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 1] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 2] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 3] = 0.0f;
      }
    }

    __syncthreads();

    // compute
    for (size_t k = 0; k < BK; ++k) {
      cnt = 0;

      // load value in registers
      for (size_t i = 0; i < TN; ++i) {
        A_reg[i] = A_s[k][i * CN + innerRowC];
      }
      for (size_t j = 0; j < TM; ++j) {
        B_reg[j] = B_s[k][j * CM + innerColC];
      }

      for (size_t i = 0; i < TN; ++i) {
        for (size_t j = 0; j < TM; ++j) {
          sums[cnt++] += A_reg[i] * B_reg[j];
        }
      }
    }
    __syncthreads();
  }

  cnt = 0;
  for (size_t innerRowCOffset = 0; innerRowCOffset < BN;
       innerRowCOffset += CN) {
    for (size_t innerColCOffset = 0; innerColCOffset < BM;
         innerColCOffset += CM) {
      if ((rowCOffset + innerRowCOffset + innerRowC) < N &&
          (colCOffset + innerColCOffset + innerColC) < M) {
        C[(rowCOffset + innerRowCOffset + innerRowC) * M + colCOffset +
          innerColCOffset + innerColC] = sums[cnt];
      }
      ++cnt;
    }
  }
}

template <const size_t BN, const size_t BK, const size_t BM, const size_t TN,
          const size_t TM>
__global__ void matmul_tiled_vector_kernel2(float *A, float *B, float *C,
                                            size_t N, size_t K, size_t M) {
  const size_t rowAOffset = blockIdx.y * BN;
  const size_t colBOffset = blockIdx.x * BM;
  const size_t rowCOffset = rowAOffset;
  const size_t colCOffset = colBOffset;

  size_t cnt = 0;
  size_t innerColA = threadIdx.x % (BK / 4);
  size_t innerRowA = threadIdx.x / (BK / 4);
  size_t innerColB = threadIdx.x % (BM / 4);
  size_t innerRowB = threadIdx.x / (BM / 4);
  size_t threadCol = threadIdx.x % (BM / TM);
  size_t threadRow = threadIdx.x / (BM / TM);

  size_t numThreads = (BN * BM) / (TN * TM);
  size_t strideA = numThreads / (BK / 4);
  size_t strideB = numThreads / (BM / 4);

  __shared__ float A_s[BK][BN];
  __shared__ float B_s[BK][BM + 5];

  float sums[TN * TM] = {0};
  float A_reg[TN], B_reg[TM];

  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BK) {

    // load data in shared memory
    for (size_t innerRowAOffset = 0; innerRowAOffset < BN;
         innerRowAOffset += strideA) {
      if ((rowAOffset + innerRowAOffset + innerRowA) < N &&
          (tileOffset + innerColA * 4) < K) {
        float4 tmp =
            *((float4 *)&A[(rowAOffset + innerRowAOffset + innerRowA) * K +
                           tileOffset + innerColA * 4]);
        // TODO fix bank conflicts
        A_s[innerColA * 4 + 0][innerRowAOffset + innerRowA] = tmp.x;
        A_s[innerColA * 4 + 1][innerRowAOffset + innerRowA] = tmp.y;
        A_s[innerColA * 4 + 2][innerRowAOffset + innerRowA] = tmp.z;
        A_s[innerColA * 4 + 3][innerRowAOffset + innerRowA] = tmp.w;
      } else {
        A_s[innerColA * 4 + 0][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 1][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 2][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 3][innerRowAOffset + innerRowA] = 0.0f;
      }
    }

    for (size_t innerRowBOffset = 0; innerRowBOffset < BK;
         innerRowBOffset += strideB) {
      if ((tileOffset + innerRowBOffset + innerRowB) < K &&
          (colBOffset + innerColB * 4) < M) {
        float4 tmp =
            *(float4 *)&B[(tileOffset + innerRowBOffset + innerRowB) * M +
                          colBOffset + innerColB * 4];
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 0] = tmp.x;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 1] = tmp.y;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 2] = tmp.z;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 3] = tmp.w;
      } else {
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 0] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 1] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 2] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 3] = 0.0f;
      }
    }

    __syncthreads();

    // compute
    for (size_t k = 0; k < BK; ++k) {
      cnt = 0;

      // load value in registers
      for (size_t i = 0; i < TN; ++i) {
        A_reg[i] = A_s[k][threadRow * TN + i];
      }
      for (size_t j = 0; j < TM; ++j) {
        B_reg[j] = B_s[k][threadCol * TM + j];
      }

      for (size_t i = 0; i < TN; ++i) {
        for (size_t j = 0; j < TM; ++j) {
          sums[cnt++] += A_reg[i] * B_reg[j];
        }
      }
    }
    __syncthreads();
  }

  cnt = 0;
  for (size_t i = 0; i < TN; ++i) {
    for (size_t j = 0; j < TM; ++j) {
      size_t innerRowC = threadRow * TN + i;
      size_t innerColC = threadCol * TM + j;
      if ((rowCOffset + innerRowC) < N && (colCOffset + innerColC) < M) {
        C[(rowCOffset + innerRowC) * M + colCOffset + innerColC] = sums[cnt];
      }
      ++cnt;
    }
  }
}

template <const size_t BN, const size_t BK, const size_t BM, const size_t TN,
          const size_t TM, const size_t WITERN, const size_t WITERM,
          const size_t WCOL>
__global__ void matmul_tiled_warptile_kernel(float *A, float *B, float *C,
                                             size_t N, size_t K, size_t M) {
  const size_t WARPSIZE = 32;
  const size_t WSUBN = WITERN * TN;
  const size_t WSUBM = WITERM * TM;
  const size_t WN = WSUBN * 32 / WCOL;
  const size_t WM = WSUBM * WCOL;

  const size_t rowAOffset = blockIdx.y * BN;
  const size_t colBOffset = blockIdx.x * BM;
  const size_t rowCOffset = rowAOffset;
  const size_t colCOffset = colBOffset;

  size_t cnt = 0;
  size_t innerColA = threadIdx.x % (BK / 4);
  size_t innerRowA = threadIdx.x / (BK / 4);
  size_t innerColB = threadIdx.x % (BM / 4);
  size_t innerRowB = threadIdx.x / (BM / 4);

  size_t laneIdx = threadIdx.x % WARPSIZE;
  size_t warpIdx = threadIdx.x / WARPSIZE;
  size_t innerWarpColOffset = (laneIdx % WCOL) * WSUBM;
  size_t innerWarpRowOffset = (laneIdx / WCOL) * WSUBN;
  size_t warpColOffset = warpIdx % (BM / WM) * WM;
  size_t warpRowOffset = warpIdx / (BM / WM) * WN;

  size_t numThreads = (BN * BM) / (WSUBN * WSUBM);
  size_t strideA = numThreads / (BK / 4);
  size_t strideB = numThreads / (BM / 4);

  __shared__ float A_s[BK][BN];
  __shared__ float B_s[BK][BM + 5];

  float sums[WSUBN * WSUBM] = {0};
  float A_reg[WSUBN], B_reg[WSUBM];

  for (size_t tileOffset = 0; tileOffset < K; tileOffset += BK) {

    // load data in shared memory
    for (size_t innerRowAOffset = 0; innerRowAOffset < BN;
         innerRowAOffset += strideA) {
      if ((rowAOffset + innerRowAOffset + innerRowA) < N &&
          (tileOffset + innerColA * 4) < K) {
        float4 tmp =
            *((float4 *)&A[(rowAOffset + innerRowAOffset + innerRowA) * K +
                           tileOffset + innerColA * 4]);
        // TODO fix bank conflicts
        A_s[innerColA * 4 + 0][innerRowAOffset + innerRowA] = tmp.x;
        A_s[innerColA * 4 + 1][innerRowAOffset + innerRowA] = tmp.y;
        A_s[innerColA * 4 + 2][innerRowAOffset + innerRowA] = tmp.z;
        A_s[innerColA * 4 + 3][innerRowAOffset + innerRowA] = tmp.w;
      } else {
        A_s[innerColA * 4 + 0][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 1][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 2][innerRowAOffset + innerRowA] = 0.0f;
        A_s[innerColA * 4 + 3][innerRowAOffset + innerRowA] = 0.0f;
      }
    }

    for (size_t innerRowBOffset = 0; innerRowBOffset < BK;
         innerRowBOffset += strideB) {
      if ((tileOffset + innerRowBOffset + innerRowB) < K &&
          (colBOffset + innerColB * 4) < M) {
        float4 tmp =
            *(float4 *)&B[(tileOffset + innerRowBOffset + innerRowB) * M +
                          colBOffset + innerColB * 4];
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 0] = tmp.x;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 1] = tmp.y;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 2] = tmp.z;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 3] = tmp.w;
      } else {
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 0] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 1] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 2] = 0.0f;
        B_s[innerRowBOffset + innerRowB][innerColB * 4 + 3] = 0.0f;
      }
    }

    __syncthreads();

    // compute
    for (size_t k = 0; k < BK; ++k) {
      cnt = 0;

      // load value in registers
      for (size_t wSubRowOffset = 0; wSubRowOffset < WSUBN;
           wSubRowOffset += TN) {
        for (size_t i = 0; i < TN; ++i) {
          A_reg[wSubRowOffset + i] =
              A_s[k][warpRowOffset + innerWarpRowOffset + wSubRowOffset + i];
        }
      }
      for (size_t wSubColOffset = 0; wSubColOffset < WSUBM;
           wSubColOffset += TM) {
        for (size_t j = 0; j < TM; ++j) {
          B_reg[wSubColOffset + j] =
              B_s[k][warpColOffset + innerWarpColOffset + wSubColOffset + j];
        }
      }

      for (size_t wSubRowOffset = 0; wSubRowOffset < WSUBN;
           wSubRowOffset += TN) {
        for (size_t wSubColOffset = 0; wSubColOffset < WSUBM;
             wSubColOffset += TM) {
          for (size_t i = 0; i < TN; ++i) {
            for (size_t j = 0; j < TM; ++j) {
              sums[cnt++] +=
                  A_reg[wSubRowOffset + i] * B_reg[wSubColOffset + j];
            }
          }
        }
      }
    }
    __syncthreads();
  }

  cnt = 0;
  for (size_t wSubRowOffset = 0; wSubRowOffset < WSUBN; wSubRowOffset += TN) {
    for (size_t wSubColOffset = 0; wSubColOffset < WSUBM; wSubColOffset += TM) {
      for (size_t i = 0; i < TN; ++i) {
        for (size_t j = 0; j < TM; ++j) {
          size_t innerRowC =
              warpRowOffset + innerWarpRowOffset + wSubRowOffset + i;
          size_t innerColC =
              warpColOffset + innerWarpColOffset + wSubColOffset + j;
          if ((rowCOffset + innerRowC) < N && (colCOffset + innerColC) < M) {
            C[(rowCOffset + innerRowC) * M + colCOffset + innerColC] =
                sums[cnt];
          }
          ++cnt;
        }
      }
    }
  }
}

bool allclose(float *A, float *B, size_t N, size_t M) {
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < M; ++j) {
      if (abs(A[i * M + j] - B[i * M + j]) > 1e-4) {
        printf("Mismatch at (%lu, %lu), A = %f and B = %f\n", i, j,
               A[i * M + j], B[i * M + j]);
        return false;
      }
    }
  }
  return true;
}

void print_mat(float *A, size_t N, size_t M) {
  printf("[\n");
  for (size_t row = 0; row < N; ++row) {
    for (size_t col = 0; col < M; ++col) {
      printf("  %f ", A[row * M + col]);
    }
    printf("\n");
  }
  printf("]\n");
}

int main() {
  size_t N, K, M;
  float *A, *B, *C_base, *C;
  float *A_d, *B_d, *C_base_d, *C_d;
  dim3 numThreads, numBlocks;

  N = K = M = 4096;

  A = (float *)malloc(N * K * sizeof(float));
  B = (float *)malloc(K * M * sizeof(float));
  C_base = (float *)malloc(N * M * sizeof(float));
  C = (float *)malloc(N * M * sizeof(float));

  hipMalloc(&A_d, N * K * sizeof(float));
  hipMalloc(&B_d, K * M * sizeof(float));
  hipMalloc(&C_base_d, N * M * sizeof(float));
  hipMalloc(&C_d, N * M * sizeof(float));

  for (size_t i = 0; i < N * K; ++i) {
    A[i] = (float)rand() / RAND_MAX;
  }

  for (size_t i = 0; i < K * M; ++i) {
    B[i] = (float)rand() / RAND_MAX;
  }

  CUDA_CHECK(hipMemcpy(A_d, A, N * K * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(B_d, B, K * M * sizeof(float), hipMemcpyHostToDevice));

  CUDA_CHECK(hipDeviceSynchronize());
  const size_t BLOCK_SIZE = 32;
  numThreads = dim3(BLOCK_SIZE * BLOCK_SIZE);
  numBlocks = dim3(cdiv(M, BLOCK_SIZE), cdiv(N, BLOCK_SIZE));
  matmul_naive_kernel<BLOCK_SIZE>
      <<<numBlocks, numThreads>>>(A_d, B_d, C_base_d, N, K, M);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(C_base, C_base_d, N * M * sizeof(float),
                        hipMemcpyDeviceToHost));

  // // naive matmul kernel
  // cudaMemset(C_d, 0, N * M * sizeof(float));
  // numThreads = dim3(BLOCK_SIZE * BLOCK_SIZE);
  // numBlocks = dim3(cdiv(M, BLOCK_SIZE), cdiv(N, BLOCK_SIZE));
  // BENCHMARK_CUDA_KERNEL("naive matmul kernel", 5, 20,
  //                       (matmul_naive_kernel<BLOCK_SIZE>
  //                        <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K,
  //                        M)));
  // CUDA_CHECK(cudaMemcpy(C, C_d, N * M * sizeof(float),
  // cudaMemcpyDeviceToHost)); printf("Match impl: %s\n\n", allclose(C_base, C,
  // N, M) ? "true" : "false");

  // // tiled matmul kernel
  // cudaMemset(C_d, 0, N * M * sizeof(float));
  // numThreads = dim3(BLOCK_SIZE * BLOCK_SIZE);
  // numBlocks = dim3(cdiv(M, BLOCK_SIZE), cdiv(N, BLOCK_SIZE));
  // BENCHMARK_CUDA_KERNEL("tiled matmul kernel", 5, 20,
  //                       (matmul_tiled_kernel<BLOCK_SIZE>
  //                        <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K,
  //                        M)));
  // CUDA_CHECK(cudaMemcpy(C, C_d, N * M * sizeof(float),
  // cudaMemcpyDeviceToHost)); printf("Match impl: %s\n\n", allclose(C_base, C,
  // N, M) ? "true" : "false");

  // // blocktiled matmul kernel
  // cudaMemset(C_d, 0, N * M * sizeof(float));
  // const size_t CN = 8;
  // const size_t CM = 8;
  // const size_t BK = 8;
  // const size_t BN = 64;
  // const size_t BM = 64;
  // assert(BN * BK >= CN * CM &&
  //        BM * BK >= CN * CM); // number of threads must be less than
  // numThreads = dim3(CN * CM);
  // numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  // BENCHMARK_CUDA_KERNEL("block tiled matmul kernel", 5, 20,
  //                       (matmul_tiled_2d_kernel<BN, BK, BM, CN, CM>
  //                        <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K,
  //                        M)));
  // CUDA_CHECK(cudaMemcpy(C, C_d, N * M * sizeof(float),
  // cudaMemcpyDeviceToHost)); printf("Match impl: %s\n\n", allclose(C_base, C,
  // N, M) ? "true" : "false");

  // // vectorized block tiled matmul kernel
  // CUDA_CHECK(cudaDeviceSynchronize());
  // cudaMemset(C_d, 0, N * M * sizeof(float));
  // const size_t CN = 8;
  // const size_t CM = 8;
  // const size_t BK = 8;
  // const size_t BN = 64;
  // const size_t BM = 64;
  // assert(BN * BK >= 4 * CN * CM &&
  //        BM * BK >= 4 * CN * CM); // number of threads must be less than
  // numThreads = dim3(CN * CM);
  // numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  // BENCHMARK_CUDA_KERNEL("vectorized block tiled matmul kernel", 5, 20,
  //                       (matmul_tiled_vector_kernel<BN, BK, BM, CN, CM>
  //                        <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K,
  //                        M)));
  // CUDA_CHECK(cudaMemcpy(C, C_d, N * M * sizeof(float),
  // cudaMemcpyDeviceToHost)); printf("Match impl: %s\n\n", allclose(C_base, C,
  // N, M) ? "true" : "false");

  // // vectorized block tiled matmul kernel
  // CUDA_CHECK(cudaDeviceSynchronize());
  // cudaMemset(C_d, 0, N * M * sizeof(float));
  // const size_t TN = 8;
  // const size_t TM = 4;
  // const size_t BK = 16;
  // const size_t BN = 128;
  // const size_t BM = 64;
  // numThreads = dim3((BN * BM) / (TN * TM));
  // numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  // BENCHMARK_CUDA_KERNEL("vectorized block tiled matmul kernel", 5, 20,
  //                       (matmul_tiled_vector_kernel2<BN, BK, BM, TN, TM>
  //                        <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K,
  //                        M)));
  // CUDA_CHECK(cudaMemcpy(C, C_d, N * M * sizeof(float),
  // cudaMemcpyDeviceToHost)); printf("Match impl: %s\n\n", allclose(C_base, C,
  // N, M) ? "true" : "false");

  // vectorized block tiled matmul kernel
  CUDA_CHECK(hipDeviceSynchronize());
  hipMemset(C_d, 0, N * M * sizeof(float));
  const size_t TN = 2;
  const size_t TM = 2;
  const size_t BK = 8;
  const size_t BN = 32;
  const size_t BM = 32;
  const size_t WITERN = 2;
  const size_t WITERM = 2;
  const size_t WCOL = 4;
  assert(BN * BM > 32 * TN * TM * WITERN * WITERM); // have atleast 1 warp
  numThreads = dim3((BN * BM) / (TN * TM * WITERN * WITERM));
  numBlocks = dim3(cdiv(M, BM), cdiv(N, BN));
  hipEventCreateKERNEL(
      "vectorized block tiled matmul kernel", 5, 20,
      (matmul_tiled_warptile_kernel<BN, BK, BM, TN, TM, WITERN, WITERN, WCOL>
       <<<numBlocks, numThreads>>>(A_d, B_d, C_d, N, K, M)));
  CUDA_CHECK(hipMemcpy(C, C_d, N * M * sizeof(float), hipMemcpyDeviceToHost));
  printf("Match impl: %s\n\n", allclose(C_base, C, N, M) ? "true" : "false");

  // print_mat(C, N, M);
  // print_mat(C_base, N, M);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_base_d);
  hipFree(C_d);
  free(A);
  free(B);
  free(C_base);
  free(C);

  return 0;
}
