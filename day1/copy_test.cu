
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

struct timer {
  struct timespec start_time, end_time;
};

void start_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->start_time);
}

void stop_timer(struct timer *t) {
  clock_gettime(CLOCK_MONOTONIC, &t->end_time);
}

double time_diff(struct timer *t) {
  double diff = (t->end_time.tv_sec - t->start_time.tv_sec) +
                (t->end_time.tv_nsec - t->start_time.tv_nsec) / 1000000000.0;
  return diff;
}

struct timer t;

int32_t main() {
  int32_t N = 100000000;

  hipDeviceSynchronize();

  int32_t *x = (int32_t *)malloc(N * sizeof(int32_t));
  for (int32_t i = 0; i < N; ++i) {
    x[i] = rand();
  }

  start_timer(&t);
  int32_t *x_h = (int32_t *)malloc(N * sizeof(int32_t));
  for (int32_t i = 0; i < N; ++i) {
    x_h[i] = x[i];
  }
  stop_timer(&t);
  printf("CPU to CPU copy time: %f\n", time_diff(&t));

  int32_t *x_d;
  start_timer(&t);
  hipMalloc(&x_d, N);
  hipMemcpy(x_d, x, N, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  stop_timer(&t);
  printf("CPU to GPU copy time: %f\n", time_diff(&t));
}
