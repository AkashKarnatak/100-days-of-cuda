
#include <hip/hip_runtime.h>
#include <stdio.h>

size_t cdiv(size_t a, size_t b) { return (a + b - 1) / b; }

__global__ void flash_attention_kernel(float *query, float *key, float *value,
                                       float *output, float *sums, float *maxes,
                                       size_t N, size_t d, size_t Br, size_t Bc,
                                       size_t Tr, size_t Tc) {
  size_t tid = threadIdx.x;
  float *Q, *K, *V, *O, *S, *m, *l;

  __shared__ float sram[11 * 1024];
  Q = &sram[0];                                // Br x d
  K = &sram[Br * d];                           // Bc x d
  V = &sram[Br * d + Bc * d];                  // Bc x d
  O = &sram[Br * d + 2 * Bc * d];              // Br x d
  l = &sram[2 * Br * d + 2 * Bc * d];          // Br
  m = &sram[2 * Br * d + 2 * Bc * d + Br];     // Br
  S = &sram[2 * Br * d + 2 * Bc * d + 2 * Br]; // Br x d

  for (size_t j = 0; j < Tc; ++j) {
    __syncthreads();

    // load Kj, Vj
    for (size_t k = 0; k < d; ++k) {
      if (j * Bc + tid < N) {
        K[tid * d + k] = key[(j * Bc + tid) * d + k];
        V[tid * d + k] = value[(j * Bc + tid) * d + k];
      } else {
        K[tid * d + k] = -INFINITY;
        V[tid * d + k] = 0;
      }
    }

    __syncthreads();

    for (size_t i = 0; i < Tr; ++i) {
      // load Q, O, m, l
      if (tid < Br) {
        for (size_t k = 0; k < d; ++k) {
          if (i * Br + tid < N) {
            Q[tid * d + k] = query[(i * Br + tid) * d + k];
            O[tid * d + k] = output[(i * Br + tid) * d + k];
          } else {
            Q[tid * d + k] = -INFINITY;
            O[tid * d + k] = 0;
          }
        }
        if (i * Br + tid < N) {
          l[tid] = sums[i * Br + tid];
          m[tid] = maxes[i * Br + tid];
        } else {
          l[tid] = 0;
          m[tid] = -INFINITY;
        }
      }

      __syncthreads();

      if (tid >= Br || i * Br + tid >= N)
        continue;

      float m_, l_, m_new, l_new;
      m_ = -INFINITY, l_ = 0;

      for (size_t col = 0; col < Bc; ++col) {
        float sum = 0.0f;
        for (size_t k = 0; k < d; ++k) {
          sum += Q[tid * d + k] * K[col * d + k];
        }
        S[tid * Bc + col] = sum / sqrtf(d);
        m_ = fmaxf(m_, sum / sqrtf(d));
      }

      for (size_t col = 0; col < Bc; ++col) {
        S[tid * Bc + col] = expf(S[tid * Bc + col] - m_);
        l_ += S[tid * Bc + col];
      }

      m_new = max(m[tid], m_);
      l_new = l[tid] * expf(m[tid] - m_new) + l_ * expf(m_ - m_new);

      for (size_t col = 0; col < d; ++col) {
        float sum = 0.0f;
        for (size_t k = 0; k < Bc; ++k) {
          sum += S[tid * Bc + k] * V[k * d + col];
        }
        output[(i * Br + tid) * d + col] =
            (l[tid] * O[tid * d + col] * expf(m[tid] - m_new) +
             sum * expf(m_ - m_new)) /
            l_new;
      }
      sums[i * Br + tid] = l_new;
      maxes[i * Br + tid] = m_new;
    }
  }
}

extern "C" {
void flash_attention_gpu(float *query, float *key, float *value, float *output,
                         float *sums, float *maxes, size_t N, size_t d) {
  size_t Br, Bc, Tr, Tc;
  float *query_d, *key_d, *value_d, *output_d, *sums_d, *maxes_d;

  Br = 16, Bc = 16;
  Tr = cdiv(N, Br), Tc = cdiv(N, Bc);

  hipMalloc(&query_d, (N * d) * sizeof(float));
  hipMalloc(&key_d, (N * d) * sizeof(float));
  hipMalloc(&value_d, (N * d) * sizeof(float));
  hipMalloc(&output_d, (N * d) * sizeof(float));
  hipMalloc(&sums_d, N * sizeof(float));
  hipMalloc(&maxes_d, N * sizeof(float));

  hipMemcpy(query_d, query, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(key_d, key, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(value_d, value, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(output_d, output, (N * d) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(sums_d, sums, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(maxes_d, maxes, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 numThreads(Bc);
  dim3 numBlocks(1);
  flash_attention_kernel<<<numBlocks, numThreads>>>(
      query_d, key_d, value_d, output_d, sums_d, maxes_d, N, d, Br, Bc, Tr, Tc);
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, (N * d) * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(sums, sums_d, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(maxes, maxes_d, N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(query_d);
  hipFree(key_d);
  hipFree(value_d);
  hipFree(output_d);
  hipFree(sums_d);
  hipFree(maxes_d);
}
}

int main() {
  hipDeviceSynchronize();

  size_t N, d;
  float *query, *key, *value, *output, *sums, *maxes;

  N = 64, d = 64;

  query = (float *)malloc((N * d) * sizeof(float));
  key = (float *)malloc((N * d) * sizeof(float));
  value = (float *)malloc((N * d) * sizeof(float));
  output = (float *)malloc((N * d) * sizeof(float));
  sums = (float *)malloc(N * sizeof(float));
  maxes = (float *)malloc(N * sizeof(float));

  for (size_t i = 0; i < N * d; ++i) {
    query[i] = (float)rand() / RAND_MAX;
    key[i] = (float)rand() / RAND_MAX;
    value[i] = (float)rand() / RAND_MAX;
    output[i] = 0;
  }

  for (size_t i = 0; i < N; ++i) {
    sums[i] = 0;
    maxes[i] = -INFINITY;
  }

  flash_attention_gpu(query, key, value, output, sums, maxes, N, d);
  hipError_t err = hipGetLastError(); // Check for launch errors
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }

  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < d; ++j) {
      printf(" %.4f", output[i * d + j]);
    }
    printf("\n");
  }

  free(query);
  free(key);
  free(value);
  free(output);
  free(sums);
  free(maxes);
}
