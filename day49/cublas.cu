#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3
#define M 4
#define K 2

void printMatrix(const float *A, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      printf("%15.5f", A[i * cols + j]);
    }
    printf("\n");
  }
}

int main() {
  float A[N * K] = {1, 2, 3, 4, 5, 6};
  float B[K * M] = {7, 8, 9, 10, 11, 12, 13, 14};
  float C[N * M];

  float *A_d, *B_d, *C_d;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipMalloc(&A_d, N * K * sizeof(float));
  hipMalloc(&B_d, K * M * sizeof(float));
  hipMalloc(&C_d, N * M * sizeof(float));

  hipblasSetMatrix(N, K, sizeof(float), A, N, A_d, N);
  hipblasSetMatrix(K, M, sizeof(float), B, K, B_d, K);

  const float alpha = 1.0f, beta = 0.0f;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, B_d, M, A_d, K,
              &beta, C_d, M);

  hipblasGetMatrix(N, M, sizeof(float), C_d, N, C, N);

  printf("Result matrix:\n");
  printMatrix(C, N, M);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipblasDestroy(handle);

  return 0;
}
